#include <stdio.h>

#include "cuda_shared.h"     // nve has it
#include "cuda_common.h"
#include "crm_cuda_utils.cu" // nve has it

#include "fix_fire_cuda_cu.h"
#include "fix_fire_cuda_kernel.cu"

#define THREADS_PER_BLOCK 256

#define CU_CHECK(x)  { hipError_t err = x; if (err != hipSuccess) return err; }


/*!
 * Launcher for array reduction
 */
template<typename T>
hipError_t reduce(int n, T *arr, T &res)
{
  int ncur = n;

  int num_blocks = ncur / THREADS_PER_BLOCK;
  if (num_blocks*THREADS_PER_BLOCK < ncur)
    num_blocks++;

  T *buf1, *buf2;
  // CudaWrapper_AllocCudaData(unsigned nbytes); // Here we allocate memory. report to developer in printf // TAS
  CU_CHECK( hipMalloc(&buf1, num_blocks*sizeof(T)) ); //printf("#CUDA Allocbuf1 %ubytes at%p\n",num_blocks*sizeof(T), buf1);
  CU_CHECK( hipMalloc(&buf2, num_blocks*sizeof(T)) );//printf("#CUDA Allocbuf2 %ubytes at%p\n",num_blocks*sizeof(T), buf2);

  /*
   * First reduction step. Now have num_blocks entries in buf2
   */
  reduce_kernel<T, THREADS_PER_BLOCK>
    <<<num_blocks, THREADS_PER_BLOCK, THREADS_PER_BLOCK*sizeof(T)>>>
    (ncur, arr, buf2);
  CU_CHECK( hipPeekAtLastError() );

  /*
   * We need to reduce these num_blocks entries
   */
  ncur = num_blocks;
  while (ncur > 1) {
    /*
     * Flip buf1 and buf2. buf2 becomes input buffer
     */
    T *tmp = buf1;
    buf1 = buf2;
    buf2 = tmp;

    num_blocks = ncur / THREADS_PER_BLOCK;
    if (num_blocks*THREADS_PER_BLOCK < ncur)
      num_blocks++;

    /*
     * Reduction step.
     */
    reduce_kernel<T, THREADS_PER_BLOCK>
      <<<num_blocks, THREADS_PER_BLOCK, THREADS_PER_BLOCK*sizeof(T)>>>
      (ncur, buf1, buf2);
    CU_CHECK( hipPeekAtLastError() );

    ncur = num_blocks;
  }
  // Here we copy to host.  Does it get reported? TAS
  CU_CHECK( hipMemcpy(&res, buf2, sizeof(T), hipMemcpyDeviceToHost) );//printf("#CUDA CP to host %u bytes\n",sizeof(T));//TAS

  CU_CHECK( hipFree(buf1) );//printf("#CUDA free buf1%p \n",buf1);//TAS
  CU_CHECK( hipFree(buf2) );//printf("#CUDA free buf2%p \n",buf2);//TAS

  return hipSuccess;
}

/*!
 * Compute the dot products v.f, v.v and f.f
 */
hipError_t fix_fire_cuda_dot_products(int groupbit, int nlocal, int nmax,
                                     int *dev_mask, double *dev_v,
                                     double *dev_f, double &vf,
                                     double &vg_dot_vg, double &Fg_dot_Fg)
{
  int num_blocks = nlocal / THREADS_PER_BLOCK;

  if (num_blocks*THREADS_PER_BLOCK < nlocal)
    num_blocks++;

  double *dev_vf, *dev_vg_dot_vg, *dev_Fg_dot_Fg;
  CU_CHECK( hipMalloc(&dev_vf, num_blocks*sizeof(double)) );//printf("#CUDA alloc%ubytes at%p \n",num_blocks*sizeof(double), dev_vf);//TAS
  CU_CHECK( hipMalloc(&dev_vg_dot_vg, num_blocks*sizeof(double)) );//printf("#CUDA alloc%ubytes at%p \n",num_blocks*sizeof(double), dev_vg_dot_vg);//TAS
  CU_CHECK( hipMalloc(&dev_Fg_dot_Fg, num_blocks*sizeof(double)) );//printf("#CUDA alloc%ubytes at%p \n",num_blocks*sizeof(double), dev_Fg_dot_Fg);//TAS

  fix_fire_cuda_dot_products<THREADS_PER_BLOCK>
      <<<num_blocks, THREADS_PER_BLOCK, 3*THREADS_PER_BLOCK*sizeof(double)>>>
    (groupbit, nlocal, nmax, dev_mask, dev_v, dev_f, dev_vf, dev_vg_dot_vg,
     dev_Fg_dot_Fg);
  CU_CHECK( hipPeekAtLastError() );

  // Reduce each thread's values and copy to host pointer
  CU_CHECK( reduce(num_blocks, dev_vf, vf) );
  CU_CHECK( reduce(num_blocks, dev_vg_dot_vg, vg_dot_vg) );
  CU_CHECK( reduce(num_blocks, dev_Fg_dot_Fg, Fg_dot_Fg) );

  CU_CHECK( hipFree(dev_vf) );//printf("#CUDA free %p \n",dev_vf);//TAS
  CU_CHECK( hipFree(dev_vg_dot_vg) );//printf("#CUDA free %p \n",dev_vg_dot_vg);//TAS
  CU_CHECK( hipFree(dev_Fg_dot_Fg) );//printf("#CUDA free %p \n",dev_Fg_dot_Fg);//TAS

  return hipSuccess;
}

/*!
 * Mix each atom's force vector into its velocity vector
 */
hipError_t fix_fire_cuda_mix(double a, double b, int groupbit, int nlocal,
                            int nmax, int *dev_mask, double *dev_v,
                            double *dev_f)
{
  int num_blocks = nlocal / THREADS_PER_BLOCK;

  if (num_blocks*THREADS_PER_BLOCK < nlocal)
    num_blocks++;

  fix_fire_cuda_mix_kernel<<<num_blocks, THREADS_PER_BLOCK>>>
    (a, b, groupbit, nlocal, nmax, dev_mask, dev_v, dev_f);
  CU_CHECK( hipPeekAtLastError() );

  return hipSuccess;
}
